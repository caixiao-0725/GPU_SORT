#include "dataInit.cuh"
#include "oneSweep.cuh"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <hip/hip_runtime.h>
#include ""
#include <numeric>

#define TIMES 20
#define sort_type 0
// ��֤�����������ˣ�
bool verify_result(const uint32_t* sorted_keys,
    const uint32_t* sorted_values,
    int n)
{
    for (int i = 1; i < n; ++i) {
        if (sorted_keys[i - 1] > sorted_keys[i]) return false;
    }
    return true;
}


int main() {

    int device_id = 0;  // Ĭ���豸 ID
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    printf("Device Name: %s\n", prop.name);
    printf("Number of SMs (Multi-Processors): %d\n", prop.multiProcessorCount);

    // �����ڴ����
    uint32_t* h_keys = new uint32_t[N];
    uint32_t* h_values = new uint32_t[N];
    uint32_t* h_sorted_keys = new uint32_t[N];
    uint32_t* h_sorted_values = new uint32_t[N];
    // �����������
    std::iota(h_values, h_values + N, 0);

    // �豸�ڴ����
    uint32_t* d_keys[2], * d_values[2];

    hipMalloc(&d_keys[0], N * sizeof(uint32_t));
    hipMalloc(&d_keys[1], N * sizeof(uint32_t));
    hipMalloc(&d_values[0], N * sizeof(uint32_t));
    hipMalloc(&d_values[1], N * sizeof(uint32_t));

    // ������ʼ���ݵ��豸
    //hipMemcpy(d_keys[0], h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_values[0], h_values, N * sizeof(uint32_t), hipMemcpyHostToDevice);

    uint32_t* index;
    uint32_t* d_histogram;

    //for cub radix sort and cub reduce
    size_t temp_storage_bytes{ 0 };
    std::shared_ptr<thrust::device_vector<float>> cub_temp_storge{ nullptr }; //temp_storge_size
    
    switch (sort_type) {
    case 0:
        hipMalloc(&index, PASS * sizeof(uint32_t));
        hipMalloc(&d_histogram, PASS * globalHistOffset * sizeof(uint32_t));
        break;
    case 1:
        cub_temp_storge = std::make_shared<thrust::device_vector<float>>(N * 4);
        hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, d_keys[0], d_keys[1], d_values[0], d_values[1], N);
        if (temp_storage_bytes > 0 && cub_temp_storge->size() < temp_storage_bytes) 
            cub_temp_storge->resize(temp_storage_bytes);        
        break;
    }
    

    // ������ʼ���ݵ��豸
    //hipMemcpy(d_keys[0], h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_values[0], h_values, N * sizeof(uint32_t), hipMemcpyHostToDevice);

    for (int time = 1;time < TIMES + 1;time++) {

        InitRandom << <256, 1024 >> > (d_keys[0], N, time);
        //InitDescending << <256, 1024 >> > (d_keys[0], N);

        switch (sort_type)
        {
        case 0:
            // ����ֱ��ͼ
            //hipMemset(d_histogram, 0, PASS * globalHistOffset * sizeof(uint32_t));
            //hipMemset(index, 0, PASS * sizeof(uint32_t));
            //hipMemcpy(d_values[0], h_values, N * sizeof(uint32_t), hipMemcpyHostToDevice);
            mem_init << <(N + 255) / 256, 256 >> > (d_values[0], d_histogram, index,
                N, PASS, PASS * globalHistOffset);

            // ����1������ֱ��ͼ
            global_histogram_kernel << <globalHistThreadblocks, globalHistThreads >> > (
                d_keys[0], d_histogram, N, 0u, 32u);

            exclusive_sum_kernel << <PASS, RADIX >> > (d_histogram);

            // ����������ѭ��
            for (int bit = 0, pass = 0; bit < 32; bit += BITS_PER_PASS, pass++) {
                int input_idx = (bit / BITS_PER_PASS) % 2;
                int output_idx = 1 - input_idx;

                digit_binning_kernel << <binningThreadblocks, binningThreads >> > (
                    d_keys[input_idx],
                    d_keys[output_idx],
                    d_values[input_idx],
                    d_values[output_idx],
                    d_histogram,
                    index,
                    N,
                    bit,
                    pass);
            }

            break;
        case 1:
            hipcub::DeviceRadixSort::SortPairs(cub_temp_storge->data().get(), temp_storage_bytes, d_keys[0], d_keys[1], d_values[0], d_values[1], N);
            break;
        }
        

    }


    // �������������
    int final_idx;
    switch (sort_type){
        case 0: 
            final_idx = (32 / BITS_PER_PASS) % 2;
            break;
        case 1: final_idx = 1;
            break;
    }
    hipMemcpy(h_sorted_keys, d_keys[final_idx], N * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_sorted_values, d_values[final_idx], N * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (int i = 0;i < 10 && i < N;i++) {
        printf("%u %u\n", h_sorted_keys[i], h_sorted_values[i]);
    }

    // ��֤���
    bool success = verify_result(h_sorted_keys, h_sorted_values, N);

    // ������
    std::cout << "===== �ֶ�ʵ������ =====" << std::endl;
    std::cout << "��֤���: " << (success ? "�ɹ�" : "ʧ��") << std::endl;

    return 0;
}